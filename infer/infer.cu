#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <NvInfer.h>
#include <opencv2/opencv.hpp>
#include <chrono>

#include "utils.h"

using namespace std;
using namespace nvinfer1;

class Logger : public ILogger
{
  void log(Severity severity, const char * msg) override
  {
    if (severity != Severity::kINFO)
      cout << msg << endl;
  }
} gLogger;

int main() {
  // https://github.com/NVIDIA-Jetson/tf_to_trt_image_classification/blob/master/examples/classify_image/classify_image.cu
  // load plan
  // https://github.com/NVIDIA-Jetson/tf_to_trt_image_classification/blob/master/examples/classify_image/classify_image.cu
  cout<<"started"<<endl;
  string project_name = "resnet_v1_50_finetuned_4class_altered_model";
  ifstream planFile("../weights/" + project_name + ".plan");
  if(!planFile.is_open()) return 1;
  stringstream planBuffer;
  planBuffer << planFile.rdbuf();
  string plan = planBuffer.str();
  IRuntime *runtime = createInferRuntime(gLogger);
  ICudaEngine *engine = runtime->deserializeCudaEngine((void*)plan.data(), plan.size(), nullptr);
  IExecutionContext *context = engine->createExecutionContext();

  int inputBindingIndex, outputBindingIndex;
  inputBindingIndex = engine->getBindingIndex("images");
  outputBindingIndex = engine->getBindingIndex("resnet_v1_50/SpatialSqueeze");
  Dims inputDims, outputDims;
  inputDims = engine->getBindingDimensions(inputBindingIndex);
  outputDims = engine->getBindingDimensions(outputBindingIndex);
  int inputWidth, inputHeight;
  inputHeight = inputDims.d[1];
  inputWidth = inputDims.d[2];


  // 画像処理, preprocessing
  cv::Mat img = cv::imread("test_images/29.jpg", CV_LOAD_IMAGE_COLOR);
  cv::cvtColor(img, img, cv::COLOR_BGR2RGB, 3);
  cv::resize(img, img, cv::Size(inputWidth, inputHeight));
  /* std::cout<<img.rows<<" "<<img.cols<<std::endl; */
  /* cv::imshow("", img); */
  /* cv::waitKey(0); */
  size_t numInput, numOutput;
  numInput = numTensorElements(inputDims);
  numOutput = numTensorElements(outputDims);
  float *inputDataHost, *outputDataHost;
  inputDataHost = (float*) malloc(numInput * sizeof(float));
  outputDataHost = (float*) malloc(numOutput * sizeof(float));
  float *inputDataDevice, *outputDataDevice;

  bool useMappedMemory = false;
  if (useMappedMemory) {
    hipHostGetDevicePointer(&inputDataDevice, inputDataHost, 0);
    hipHostGetDevicePointer(&outputDataDevice, outputDataHost, 0);
  }else {
    hipMalloc(&inputDataDevice, numInput * sizeof(float));
    hipMalloc(&outputDataDevice, numOutput * sizeof(float));
  }
  void *bindings[2];
  cvImageToTensor(img, inputDataHost, inputDims);
  preprocessVgg(inputDataHost, inputDims);
  bindings[inputBindingIndex] = (void*) inputDataDevice;
  bindings[outputBindingIndex] = (void*) outputDataDevice;

  for(int t=0; t<10; t++) {
    auto t0 = chrono::steady_clock::now();
    if (useMappedMemory) {
      // 実行
      context->execute(1, bindings);
      cout<<outputDataHost[0]<<endl;
    } else {
      // 画像のコピー
      hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
      // 実行
      context->execute(1, bindings);
      // 結果のコピー
      hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);
      cout<<outputDataHost[0]<<endl;
    }
    auto t1 = chrono::steady_clock::now();
    cout<<chrono::duration_cast<chrono::milliseconds>(t1 - t0).count()<<"ms"<<endl;
  }

  /*
  for(int i=0; i<numOutput; i++){
    cout<<i<<":"<<outputDataHost[i]<<endl;
  }
  */

  runtime->destroy();
  engine->destroy();
  context->destroy();
  free(inputDataHost);
  free(outputDataHost);
  hipFree(inputDataDevice);
  hipFree(outputDataDevice);

  return 0;
}
